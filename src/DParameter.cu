#include "hip/hip_runtime.h"
#include "DParameter.h"
#include "ChemData.h"

cfd::DParameter::DParameter(cfd::Parameter &parameter, Species &species, Reaction &reaction) :
    myid{parameter.get_int("myid")}, inviscid_scheme{parameter.get_int("inviscid_scheme")},
    reconstruction{parameter.get_int("reconstruction")}, limiter{parameter.get_int("limiter")},
    viscous_scheme{parameter.get_int("viscous_order")}, rans_model{parameter.get_int("RANS_model")},
    turb_implicit{parameter.get_int("turb_implicit")}, chemSrcMethod{parameter.get_int("chemSrcMethod")},
    Pr(parameter.get_real("prandtl_number")), cfl(parameter.get_real("cfl")),
    Prt(parameter.get_real("turbulent_prandtl_number")), Sct(parameter.get_real("turbulent_schmidt_number")) {
  const auto &spec = species;
  n_spec = spec.n_spec;
  n_scalar = parameter.get_int("n_scalar");
  auto mem_sz = n_spec * sizeof(real);
  hipMalloc(&mw, mem_sz);
  hipMemcpy(mw, spec.mw.data(), mem_sz, hipMemcpyHostToDevice);
  high_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(high_temp_coeff.data(), spec.high_temp_coeff.data(), high_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  low_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(low_temp_coeff.data(), spec.low_temp_coeff.data(), low_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  hipMalloc(&t_low, mem_sz);
  hipMalloc(&t_mid, mem_sz);
  hipMalloc(&t_high, mem_sz);
  hipMemcpy(t_low, spec.t_low.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_mid, spec.t_mid.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_high, spec.t_high.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&LJ_potent_inv, mem_sz);
  hipMemcpy(LJ_potent_inv, spec.LJ_potent_inv.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&vis_coeff, mem_sz);
  hipMemcpy(vis_coeff, spec.vis_coeff.data(), mem_sz, hipMemcpyHostToDevice);
  WjDivWi_to_One4th.init_with_size(n_spec, n_spec);
  hipMemcpy(WjDivWi_to_One4th.data(), spec.WjDivWi_to_One4th.data(), WjDivWi_to_One4th.size() * sizeof(real),
             hipMemcpyHostToDevice);
  sqrt_WiDivWjPl1Mul8.init_with_size(n_spec, n_spec);
  hipMemcpy(sqrt_WiDivWjPl1Mul8.data(), spec.sqrt_WiDivWjPl1Mul8.data(),
             sqrt_WiDivWjPl1Mul8.size() * sizeof(real), hipMemcpyHostToDevice);
  Sc = parameter.get_real("schmidt_number");

  memset(limit_flow.ll, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.ul, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.sv_inf, 0, sizeof(real) * (MAX_SPEC_NUMBER + 2));
  // density limits
  limit_flow.ll[0] = 1e-6 * parameter.get_real("rho_inf");
  limit_flow.ul[0] = 1e+3 * parameter.get_real("rho_inf");
  for (integer l = 1; l < 4; ++l) {
    // u,v,w
    limit_flow.ll[l] = -1e+3 * parameter.get_real("v_inf");
    limit_flow.ul[l] = 1e+3 * parameter.get_real("v_inf");
  }
  // pressure limits
  limit_flow.ll[4] = 1e-6 * parameter.get_real("p_inf");
  limit_flow.ul[4] = 1e+3 * parameter.get_real("p_inf");
  if (rans_model == 2) {
    // SST model
    limit_flow.ul[5] = std::numeric_limits<real>::max();
    limit_flow.ul[6] = std::numeric_limits<real>::max();
  }
  auto &sv_inf{parameter.get_real_array("sv_inf")};
  for (integer l = 0; l < n_scalar; ++l) {
    limit_flow.sv_inf[l] = sv_inf[l];
  }
}
