#include "Thermo.cuh"
#include "DParameter.h"
#include "Constants.h"

__device__ void cfd::compute_enthalpy(real t, real *enthalpy, cfd::DParameter *param) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t}, t5{t4 * t};
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->t_low[i]) {
      const real tt = param->t_low[i];
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      auto &coeff = param->low_temp_coeff;
      enthalpy[i] = coeff(i, 0) * tt + 0.5 * coeff(i, 1) * tt2 + coeff(i, 2) * tt3 / 3 + 0.25 * coeff(i, 3) * tt4 +
                    0.2 * coeff(i, 4) * tt5 + coeff(i, 5);
      const real cp = coeff(i, 0) + coeff(i, 1) * tt + coeff(i, 2) * tt2 + coeff(i, 3) * tt3 + coeff(i, 4) * tt4;
      enthalpy[i] += cp * (t - tt); // Do a linear interpolation for enthalpy
    } else {
      auto &coeff = t < param->t_mid[i] ? param->low_temp_coeff : param->high_temp_coeff;
      enthalpy[i] = coeff(i, 0) * t + 0.5 * coeff(i, 1) * t2 + coeff(i, 2) * t3 / 3 + 0.25 * coeff(i, 3) * t4 +
                    0.2 * coeff(i, 4) * t5 + coeff(i, 5);
    }
    enthalpy[i] *= cfd::R_u / param->mw[i];
  }
}

__device__ void cfd::compute_enthalpy_and_cp(real t, real *enthalpy, real *cp, const DParameter *param) {
  const double t2{t * t}, t3{t2 * t}, t4{t3 * t}, t5{t4 * t};
  for (int i = 0; i < param->n_spec; ++i) {
    if (t < param->t_low[i]) {
      const double tt = param->t_low[i];
      const double tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt, tt5 = tt4 * tt;
      auto &coeff = param->low_temp_coeff;
      enthalpy[i] = coeff(i, 0) * tt + 0.5 * coeff(i, 1) * tt2 + coeff(i, 2) * tt3 / 3 + 0.25 * coeff(i, 3) * tt4 +
                    0.2 * coeff(i, 4) * tt5 + coeff(i, 5);
      cp[i] = coeff(i, 0) + coeff(i, 1) * tt + coeff(i, 2) * tt2 + coeff(i, 3) * tt3 + coeff(i, 4) * tt4;
      enthalpy[i] += cp[i] * (t - tt); // Do a linear interpolation for enthalpy
    } else {
      auto &coeff = t < param->t_mid[i] ? param->low_temp_coeff : param->high_temp_coeff;
      enthalpy[i] = coeff(i, 0) * t + 0.5 * coeff(i, 1) * t2 + coeff(i, 2) * t3 / 3 + 0.25 * coeff(i, 3) * t4 +
                    0.2 * coeff(i, 4) * t5 + coeff(i, 5);
      cp[i] = coeff(i, 0) + coeff(i, 1) * t + coeff(i, 2) * t2 + coeff(i, 3) * t3 + coeff(i, 4) * t4;
    }
    enthalpy[i] *= R_u / param->mw[i];
    cp[i] *= R_u / param->mw[i];
  }
}

__device__ void cfd::compute_cp(real t, real *cp, cfd::DParameter *param) {
  const real t2{t * t}, t3{t2 * t}, t4{t3 * t};
  for (auto i = 0; i < param->n_spec; ++i) {
    if (t < param->t_low[i]) {
      const real tt = param->t_low[i];
      const real tt2 = tt * tt, tt3 = tt2 * tt, tt4 = tt3 * tt;
      auto &coeff = param->low_temp_coeff;
      cp[i] = coeff(i, 0) + coeff(i, 1) * tt + coeff(i, 2) * tt2 + coeff(i, 3) * tt3 + coeff(i, 4) * tt4;
    } else {
      auto &coeff = t < param->t_mid[i] ? param->low_temp_coeff : param->high_temp_coeff;
      cp[i] = coeff(i, 0) + coeff(i, 1) * t + coeff(i, 2) * t2 + coeff(i, 3) * t3 + coeff(i, 4) * t4;
    }
    cp[i] *= R_u / param->mw[i];
  }
}