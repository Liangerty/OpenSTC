#include "Parallel.h"
#include <cstdio>

void cfd::MpiParallel::setup_gpu_device() const {
  int deviceCount{0};
  hipGetDeviceCount(&deviceCount);

  if (deviceCount < n_proc) {
    printf("Not enough GPU devices.\n"
           "We want %d GPUs but only %d GPUs are available.\n"
           " Stop computing.\n", n_proc, deviceCount);
    exit();
  }

  hipDeviceProp_t prop{};
  hipGetDeviceProperties(&prop,my_id);
  hipSetDevice(my_id);
  printf("Process %d will compute on device %s.\n", my_id, prop.name);
}
