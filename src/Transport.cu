#include "hip/hip_runtime.h"
#include "Transport.cuh"
#include "DParameter.h"
#include "Field.h"
#include "Constants.h"

__host__ __device__
real cfd::Sutherland(real temperature) {
  return 1.716e-5 * pow(temperature / 273, 1.5) * (273 + 111) / (temperature + 111);
}

real cfd::compute_viscosity(real temperature, real mw_total, real const *Y, Species &spec) {
  // This method can only be used on CPU, while for GPU the allocation may be performed in every step
  for (int i = 0; i < spec.n_spec; ++i) {
    spec.x[i] = Y[i] * mw_total / spec.mw[i];
    const real t_dl{temperature * spec.LJ_potent_inv[i]};  // dimensionless temperature
    const real collision_integral{1.147 * std::pow(t_dl, -0.145) + std::pow(t_dl + 0.5, -2)};
    spec.vis_spec[i] = spec.vis_coeff[i] * std::sqrt(temperature) / collision_integral;
  }
  for (int i = 0; i < spec.n_spec; ++i) {
    for (int j = 0; j < spec.n_spec; ++j) {
      if (i == j) {
        spec.partition_fun(i, j) = 1.0;
      } else {
        const real numerator{1 + std::sqrt(spec.vis_spec[i] / spec.vis_spec[j]) * spec.WjDivWi_to_One4th(i, j)};
        spec.partition_fun(i, j) = numerator * numerator * spec.sqrt_WiDivWjPl1Mul8(i, j);
      }
    }
  }
  real viscosity{0};
  for (int i = 0; i < spec.n_spec; ++i) {
    real vis_temp{0};
    for (int j = 0; j < spec.n_spec; ++j) {
      vis_temp += spec.partition_fun(i, j) * spec.x[j];
    }
    viscosity += spec.vis_spec[i] * spec.x[i] / vis_temp;
  }
  return viscosity;
}

__device__ void
cfd::compute_transport_property(integer i, integer j, integer k, real temperature, real mw_total, const real *cp,
                                cfd::DParameter *param, DZone *zone) {
  const auto n_spec{param->n_spec};
  const real *mw = param->mw;
  const auto yk = zone->sv;

  real X[MAX_SPEC_NUMBER], vis[MAX_SPEC_NUMBER];
  for (int l = 0; l < n_spec; ++l) {
    X[l] = yk(i, j, k, l) * mw_total / mw[l];
    const real t_dl{temperature * param->LJ_potent_inv[l]}; //dimensionless temperature
    const real collision_integral{1.147 * std::pow(t_dl, -0.145) + std::pow(t_dl + 0.5, -2)};
    vis[l] = param->vis_coeff[l] * std::sqrt(temperature) / collision_integral;
  }

  real viscosity = 0;
  real conductivity = 0;
  for (int m = 0; m < n_spec; ++m) {
    real vis_temp{0};
    for (int n = 0; n < n_spec; ++n) {
      real partition_func{1.0};
      if (m != n) {
        const real numerator{1 + std::sqrt(vis[m] / vis[n]) * param->WjDivWi_to_One4th(m, n)};
        partition_func = numerator * numerator * param->sqrt_WiDivWjPl1Mul8(m, n);
      }
      vis_temp += partition_func * X[n];
    }
    const real cond_temp = 1.065 * vis_temp - 0.065 * X[m];
    viscosity += vis[m] * X[m] / vis_temp;
    const real lambda = vis[m] * (cp[m] + 1.25 * R_u / mw[m]);
    conductivity += lambda * X[m] / cond_temp;
  }
  zone->mul(i, j, k) = viscosity;
  zone->thermal_conductivity(i, j, k) = conductivity;

  // The diffusivity is now computed via constant Schmidt number method
  const real sc{param->Sc};
  for (auto l = 0; l < n_spec; ++l) {
    if (std::abs(X[l] - 1) < 1e-3) {
      zone->rho_D(i, j, k, l) = viscosity / sc;
    } else {
      zone->rho_D(i, j, k, l) = (1 - yk(i, j, k, l)) * viscosity / ((1 - X[l]) * sc);
    }
  }
}

__device__ real
cfd::compute_viscosity(integer i, integer j, integer k, real temperature, real mw_total, cfd::DParameter *param,
                       DZone *zone) {
  const auto n_spec{param->n_spec};
  const real *mw = param->mw;
  const auto &yk = zone->sv;

  real X[MAX_SPEC_NUMBER], vis[MAX_SPEC_NUMBER];
  for (int l = 0; l < n_spec; ++l) {
    X[l] = yk(i, j, k, l) * mw_total / mw[l];
    const real t_dl{temperature * param->LJ_potent_inv[l]}; //dimensionless temperature
    const real collision_integral{1.147 * std::pow(t_dl, -0.145) + std::pow(t_dl + 0.5, -2)};
    vis[l] = param->vis_coeff[l] * std::sqrt(temperature) / collision_integral;
  }

  real viscosity = 0;
  for (int m = 0; m < n_spec; ++m) {
    real vis_temp{0};
    for (int n = 0; n < n_spec; ++n) {
      real partition_func{1.0};
      if (m != n) {
        const real numerator{1 + std::sqrt(vis[m] / vis[n]) * param->WjDivWi_to_One4th(m, n)};
        partition_func = numerator * numerator * param->sqrt_WiDivWjPl1Mul8(m, n);
      }
      vis_temp += partition_func * X[n];
    }
    viscosity += vis[m] * X[m] / vis_temp;
  }
  return viscosity;
}
