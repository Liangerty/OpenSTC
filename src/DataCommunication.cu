#include "hip/hip_runtime.h"
#include "DataCommunication.cuh"
#include "Field.h"

__global__ void cfd::setup_data_to_be_sent(cfd::DZone *zone, integer i_face, real *data) {
  const auto &f = zone->parface[i_face];
  integer n[3];
  n[0] = blockIdx.x * blockDim.x + threadIdx.x;
  n[1] = blockDim.y * blockIdx.y + threadIdx.y;
  n[2] = blockIdx.z * blockDim.z + threadIdx.z;
  if (n[0] >= f.n_point[0] || n[1] >= f.n_point[1] || n[2] >= f.n_point[2]) return;

  integer idx[3];
  for (int ijk: f.loop_order) {
    idx[ijk] = f.range_start[ijk] + n[ijk] * f.loop_dir[ijk];
  }

  const integer n_var{zone->n_var}, ngg{zone->ngg};
  integer bias = n_var * (ngg + 1) * (n[f.loop_order[1]] * f.n_point[f.loop_order[2]] + n[f.loop_order[2]]);

  const auto &cv = zone->cv;
  for (integer l = 0; l < n_var; ++l) {
    data[bias + l] = cv(idx[0], idx[1], idx[2], l);
  }

  for (integer ig = 1; ig <= ngg; ++ig) {
    idx[f.face] -= f.direction;
    bias += n_var;
    for (integer l = 0; l < n_var; ++l) {
      data[bias + l] = cv(idx[0], idx[1], idx[2], l);
    }
  }
}

