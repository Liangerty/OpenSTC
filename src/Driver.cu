#include "hip/hip_runtime.h"
#include "Driver.cuh"
#include "ViscousScheme.cuh"
#include "FieldOperation.cuh"
#include "TimeAdvanceFunc.cuh"
#include "DataCommunication.cuh"
#include "Initialize.cuh"
#include "SchemeSelector.cuh"
#include <filesystem>
#include "Parallel.h"
#include <iostream>

namespace cfd {
// Instantiate all possible drivers
template
struct Driver<MixtureModel::Air, TurbMethod::Laminar>;
template
struct Driver<MixtureModel::Air, TurbMethod::RANS>;
template
struct Driver<MixtureModel::Mixture, TurbMethod::Laminar>;
template
struct Driver<MixtureModel::Mixture, TurbMethod::RANS>;
template
struct Driver<MixtureModel::FR, TurbMethod::Laminar>;
template
struct Driver<MixtureModel::FR, TurbMethod::RANS>;


template<MixtureModel mix_model, TurbMethod turb_method>
Driver<mix_model, turb_method>::Driver(Parameter &parameter, Mesh &mesh_):myid(parameter.get_int("myid")), time(),
                                                                          mesh(mesh_), parameter(parameter),
                                                                          spec(parameter), reac(parameter),
                                                                          output(myid, mesh_, field, parameter, spec) {
  // Allocate the memory for every block
  for (integer blk = 0; blk < mesh.n_block; ++blk) {
    field.emplace_back(parameter, mesh[blk]);
  }

  initialize_basic_variables(parameter, mesh, field, spec);

#ifdef GPU
  DParameter d_param(parameter, spec, reac);
  hipMalloc(&param, sizeof(DParameter));
  hipMemcpy(param, &d_param, sizeof(DParameter), hipMemcpyHostToDevice);
  for (integer blk = 0; blk < mesh.n_block; ++blk) {
    field[blk].setup_device_memory(parameter);
  }
  bound_cond.initialize_bc_on_GPU(mesh_, field, spec, parameter);
#endif
}

template<MixtureModel mix_model, TurbMethod turb_method>
void Driver<mix_model, turb_method>::initialize_computation() {
  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }
  const auto ng_1 = 2 * mesh[0].ngg - 1;

  // First, compute the conservative variables from basic variables
  for (auto i = 0; i < mesh.n_block; ++i) {
    integer mx{mesh[i].mx}, my{mesh[i].my}, mz{mesh[i].mz};
    dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
    compute_cv_from_bv<mix_model, turb_method><<<bpg, tpb>>>(field[i].d_ptr, param);
  }

  // Second, apply boundary conditions to all boundaries, including face communication between faces
  for (integer b = 0; b < mesh.n_block; ++b) {
    bound_cond.apply_boundary_conditions(mesh[b], field[b], param);
//  hipDeviceSynchronize();
    if (myid == 0) {
      printf("Boundary conditions are applied successfully for initialization\n");
    }
  }

  // Third, communicate values between processes
  data_communication<mix_model, turb_method>(mesh, field);
  // Currently not implemented, thus the current program can only be used on a single GPU

  if (myid == 0) {
    printf("Finish data transfer.\n");
  }

  for (auto b = 0; b < mesh.n_block; ++b) {
    integer mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    dim3 bpg{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
    update_physical_properties<mix_model, turb_method><<<bpg, tpb>>>(field[b].d_ptr, param);
  }
  hipDeviceSynchronize();
  if (myid == 0) {
    printf("The flowfield is completely initialized on GPU.\n");
  }
}

template<MixtureModel mix_model, TurbMethod turb_method>
void Driver<mix_model, turb_method>::simulate() {
  const auto steady{parameter.get_bool("steady")};
  if (steady) {
    steady_simulation();
  } else {
    const auto temporal_tag{parameter.get_int("temporal_scheme")};
    switch (temporal_tag) {
      case 11: // For example, if DULUSGS, then add a function to initiate the computation instead of initialize before setting up the scheme as CPU code
        break;
      case 12:break;
      default:printf("Not implemented");
    }
  }
}

template<MixtureModel mix_model, TurbMethod turb_method>
void Driver<mix_model, turb_method>::steady_simulation() {
  printf("Steady flow simulation.\n");
  bool converged{false};
  integer step{parameter.get_int("step")};
  integer total_step{parameter.get_int("total_step") + step};
  const integer n_block{mesh.n_block};
  const integer n_var{parameter.get_int("n_var")};
  const integer ngg{mesh[0].ngg};
  const integer ng_1 = 2 * ngg - 1;
  const integer output_screen = parameter.get_int("output_screen");
  const integer output_file = parameter.get_int("output_file");

  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }
  dim3 *bpg = new dim3[n_block];
  for (integer b = 0; b < n_block; ++b) {
    const auto mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    bpg[b] = {(mx - 1) / tpb.x + 1, (my - 1) / tpb.y + 1, (mz - 1) / tpb.z + 1};
  }

  while (!converged) {
    ++step;
    /*[[unlikely]]*/if (step > total_step) {
      break;
    }

    // Start a single iteration
    // First, store the value of last step
    if (step % output_screen == 0) {
      for (auto b = 0; b < n_block; ++b) {
        store_last_step <<<bpg[b], tpb >>>(field[b].d_ptr);
      }
    }

    for (auto b = 0; b < n_block; ++b) {
      set_dq_to_0 <<<bpg[b], tpb >>>(field[b].d_ptr);

      // Second, for each block, compute the residual dq
      compute_inviscid_flux<mix_model, turb_method>(mesh[b], field[b].d_ptr, param, n_var);
      compute_viscous_flux<mix_model, turb_method>(mesh[b], field[b].d_ptr, param, n_var);

      // compute local time step
      local_time_step<mix_model><<<bpg[b], tpb>>>(field[b].d_ptr, param);
      // implicit treatment if needed

      // update conservative and basic variables
      update_cv_and_bv<mix_model, turb_method><<<bpg[b], tpb>>>(field[b].d_ptr, param);

      // apply boundary conditions
      bound_cond.apply_boundary_conditions(mesh[b], field[b], param);
    }
    // Third, transfer data between and within processes
    data_communication(mesh, field);

    if (mesh.dimension == 2) {
      for (auto b = 0; b < n_block; ++b) {
        const auto mx{mesh[b].mx}, my{mesh[b].my};
        dim3 BPG{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, 1};
        eliminate_k_gradient<<<BPG, tpb>>>(field[b].d_ptr);
      }
    }

    // update physical properties such as Mach number, transport coefficients et, al.
    for (auto b = 0; b < n_block; ++b) {
      integer mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
      dim3 BPG{(mx + ng_1) / tpb.x + 1, (my + ng_1) / tpb.y + 1, (mz + ng_1) / tpb.z + 1};
      update_physical_properties<mix_model, turb_method><<<BPG, tpb>>>(field[b].d_ptr, param);
    }

    // Finally, test if the simulation reaches convergence state
    if (step % output_screen == 0) {
      real err_max = compute_residual(step);
      converged = err_max < parameter.get_real("convergence_criteria");
      if (myid == 0) {
        steady_screen_output(step, err_max);
      }
    }
    hipDeviceSynchronize();
    if (step % output_file == 0) {
      output.print_field(step);
    }
  }
  delete[] bpg;
}

template<MixtureModel mix_model, TurbMethod turb_method>
real Driver<mix_model, turb_method>::compute_residual(integer step) {
  const integer n_block{mesh.n_block};
  for (auto &e: res) {
    e = 0;
  }

  dim3 tpb{8, 8, 4};
  if (mesh.dimension == 2) {
    tpb = {16, 16, 1};
  }
  for (integer b = 0; b < n_block; ++b) {
    const auto mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    dim3 bpg = {(mx - 1) / tpb.x + 1, (my - 1) / tpb.y + 1, (mz - 1) / tpb.z + 1};
    // compute the square of the difference of the basic variables
    compute_square_of_dbv<<<bpg, tpb>>>(field[b].d_ptr);
  }

  constexpr integer TPB{128};
  constexpr integer n_res_var{4};
  real res_block[n_res_var];
  int num_sms, num_blocks_per_sm;
  hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_of_dv_squared<n_res_var>, TPB,
                                                TPB * sizeof(real) * n_res_var);
  for (integer b = 0; b < n_block; ++b) {
    const auto mx{mesh[b].mx}, my{mesh[b].my}, mz{mesh[b].mz};
    const integer size = mx * my * mz;
    int n_blocks = std::min(num_blocks_per_sm * num_sms, (size + TPB - 1) / TPB);
    reduction_of_dv_squared<n_res_var> <<<n_blocks, TPB, TPB * sizeof(real) * n_res_var >>>(
        field[b].h_ptr->bv_last.data(), size);
    reduction_of_dv_squared<n_res_var> <<<1, TPB, TPB * sizeof(real) * n_res_var >>>(field[b].h_ptr->bv_last.data(),
                                                                                     n_blocks);
    hipMemcpy(res_block, field[b].h_ptr->bv_last.data(), n_res_var * sizeof(real), hipMemcpyDeviceToHost);
    for (integer l = 0; l < n_res_var; ++l) {
      res[l] += res_block[l];
    }
  }

  if (parameter.get_bool("parallel")) {
    // Parallel reduction
  }
  for (auto &e: res) {
    e = std::sqrt(e / mesh.n_grid_total);
  }

  if (step == parameter.get_int("output_screen")) {
    for (integer i = 0; i < n_res_var; ++i) {
      res_scale[i] = res[i];
      if (res_scale[i] < 1e-20) {
        res_scale[i] = 1e-20;
      }
    }
    const std::filesystem::path out_dir("output/message");
    if (!exists(out_dir)) {
      create_directories(out_dir);
    }
    std::ofstream res_scale_out(out_dir.string() + "/residual_scale.txt");
    res_scale_out << std::format("{}\n{}\n{}\n{}\n", res_scale[0], res_scale[1], res_scale[2], res_scale[3]);
    res_scale_out.close();
  }

  for (integer i = 0; i < 4; ++i) {
    res[i] /= res_scale[i];
  }

  // Find the maximum error of the 4 errors
  real err_max = res[0];
  for (integer i = 1; i < 4; ++i) {
    if (res[i] > err_max) {
      err_max = res[i];
    }
  }

  if (myid == 0) {
    if (isnan(err_max)) {
      printf("Nan occurred in step %d. Stop simulation.\n", step);
      cfd::MpiParallel::exit();
    }
  }

  return err_max;
}

template<MixtureModel mix_model, TurbMethod turb_method>
void Driver<mix_model, turb_method>::steady_screen_output(integer step, real err_max) {
  time.get_elapsed_time();
  std::ofstream history("history.dat", std::ios::app);
  history << std::format("{}\t{}\n", step, err_max);
  history.close();

  std::cout << std::format("\n{:>38}    converged to: {:>11.4e}\n", "rho", res[0]);
  std::cout << std::format("  n={:>8},                       V     converged to: {:>11.4e}   \n", step, res[1]);
  std::cout << std::format("  n={:>8},                       p     converged to: {:>11.4e}   \n", step, res[2]);
  std::cout << std::format("{:>38}    converged to: {:>11.4e}\n", "T ", res[3]);
  std::cout << std::format("CPU time for this step is {:>16.8f}s\n", time.step_time);
  std::cout << std::format("Total elapsed CPU time is {:>16.8f}s\n", time.elapsed_time);
}

template<integer N>
__global__ void reduction_of_dv_squared(real *arr, integer size) {
  integer i = blockDim.x * blockIdx.x + threadIdx.x;
  const integer t = threadIdx.x;
  extern __shared__ real s[];
  memset(&s[t * N], 0, N * sizeof(real));
  if (i >= size) {
    return;
  }
  real inp[N];
  memset(inp, 0, N * sizeof(real));
  for (integer idx = i; idx < size; idx += blockDim.x * gridDim.x) {
    inp[0] += arr[idx];
    inp[1] += arr[idx + size];
    inp[2] += arr[idx + size * 2];
    inp[3] += arr[idx + size * 3];
  }
  for (integer l = 0; l < N; ++l) {
    s[t * N + l] = inp[l];
  }
  __syncthreads();

  for (int stride = blockDim.x / 2, lst = blockDim.x & 1; stride >= 1; lst = stride & 1, stride >>= 1) {
    stride += lst;
    __syncthreads();
    if (t < stride) {
      //when t+stride is larger than #elements, there's no meaning of comparison. So when it happens, just keep the current value for parMax[t]. This always happens when an odd number of t satisfying the condition.
      if (t + stride < size) {
        for (integer l = 0; l < N; ++l) {
          s[t * N + l] += s[(t + stride) * N + l];
        }
      }
    }
    __syncthreads();
  }

  if (t == 0) {
    arr[blockIdx.x] = s[0];
    arr[blockIdx.x + gridDim.x] = s[1];
    arr[blockIdx.x + gridDim.x * 2] = s[2];
    arr[blockIdx.x + gridDim.x * 3] = s[3];
  }
}

} // cfd